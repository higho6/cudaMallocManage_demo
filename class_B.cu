#include "hip/hip_runtime.h"
#include "class_B.h"
#include <>

__global__ void Fill(const int num,float *a,float *b)
{
    int tid = threadIdx.x;
    
    // float param[3];
    // param[0] = 1.f;
    // param[1] = 2.f;
    // param[2] = 3.f;

    for(int i=tid;i<num;i+=blockDim.x)
    {
        // for(int j=0;j<3;j++)
        // {
        //     a[i]+=param[j];
        // }
        b[i] = a[i]+b[i];
    }

}

void B::TestForward()
{
    float *dev_array_zero;
    float *dev_array_one;

    int num = 1000000;
    hipMalloc(&dev_array_zero,num*sizeof(float));
    hipMalloc(&dev_array_one,num*sizeof(float));

    hipMemset(dev_array_zero,0,num*sizeof(float));
    hipMemset(dev_array_one,0,num*sizeof(float));

    dim3 grid(1);
    dim3 block(512);
    Fill<<<grid,block>>>(num,dev_array_zero,dev_array_one);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    std::cout<<"error: "<<hipGetErrorName(err)<<std::endl;

    hipFree(dev_array_zero);
    hipFree(dev_array_one);
}