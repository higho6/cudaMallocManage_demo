#include "hip/hip_runtime.h"
#include "class_A.h"
#include <>

__global__ void Add(u_int *size)
{
    int tid = threadIdx.x;
    for(int i=tid;i<1000;i+=blockDim.x)
    {
        atomicAdd(&size[0],1);
    }
}


void A::TestForward()
{
    hipMemset(size_,0,sizeof(u_int));
    dim3 grid(1);
    dim3 block(512);
    Add<<<grid,block>>>(size_);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    std::cout<<"error: "<<hipGetErrorName(err)<<std::endl;
    std::cout<<"size: "<<size_[0]<<std::endl;
}